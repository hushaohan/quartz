#include "distributed_simulator.h"
#include "cuda_helper.h"

using namespace sim;
using namespace Legion;

DistributedSimulator::DSHandler
DistributedSimulator::cuda_init_task(Task const *task,
                                     std::vector<PhysicalRegion> const &regions,
                                     Context ctx, Runtime *runtime) {
  DSConfig const *config = (DSConfig *)task->args;
  DSHandler handle;
  handle.workSpaceSize = (size_t)1 * 1024 * 1024 * 1024; // 1GB work space
  handle.num_local_qubits = config->num_local_qubits;
  printf("Num_local_qubits = %lld\n", handle.num_local_qubits);
  custatevecCreate(&handle.statevec);
  {
    // allocate memory for workspace
    Memory gpu_mem = Machine::MemoryQuery(Machine::get_machine())
                         .only_kind(Memory::GPU_FB_MEM)
                         .best_affinity_to(task->target_proc)
                         .first();
    Realm::Rect<1, coord_t> bounds(
        Realm::Point<1, coord_t>(0),
        Realm::Point<1, coord_t>(handle.workSpaceSize - 1));
    std::vector<size_t> field_sizes;
    field_sizes.push_back(sizeof(char));
    Realm::RegionInstance workspaceInst;
    Realm::RegionInstance::create_instance(workspaceInst, gpu_mem, bounds,
                                           field_sizes, 0,
                                           Realm::ProfilingRequestSet())
        .wait();
    handle.workSpace = workspaceInst.pointer_untyped(0, sizeof(char));
  }
  handle.ncclComm = nullptr;
  return handle;
}

void DistributedSimulator::sv_init_task(
    Task const *task, std::vector<PhysicalRegion> const &regions, Context ctx,
    Runtime *runtime) {
  // TODO: implement this function
}

void DistributedSimulator::sv_comp_task(
    Task const *task, std::vector<PhysicalRegion> const &regions, Context ctx,
    Runtime *runtime) {
  DSHandler const *handler = *((DSHandler **)task->local_args);
  GateInfo const *info = (GateInfo *)task->args;
  assert(handler->vecDataType == DT_FLOAT || handler->vecDataType == DT_DOUBLE);
  hipDataType data_type = handler->vecDataType == DT_FLOAT ? HIP_C_32F : HIP_C_64F;
  custatevecComputeType_t compute_type = handler->vecDataType == DT_FLOAT ? CUSTATEVEC_COMPUTE_32F : CUSTATEVEC_COMPUTE_64F;
  GenericTensorAccessorW state_vector = helperGetGenericTensorAccessorWO(
      handler->vecDataType, regions[0], task->regions[0], FID_DATA, ctx, runtime);

  // TODO: get target & control qubit idx from current perm[]
  std::vector<int> targets;
  std::vector<int> controls;

  unsigned const nIndexBits = handler->num_local_qubits;
  unsigned const nTargets = info->num_targets;
  unsigned const nControls = info->num_controls;
  int const adjoint = 0;
  // TODO: check if targets should be ordered
  printf("Targets: [");
  for (int i = 0; i < info->num_targets; i++) {
    int idx = 0;
    while (info->permutation[idx] != info->target[i])
      idx++;
    targets.push_back(idx);
    printf("(%d, %d) ", info->target[i], idx);
  }
  printf("]\n");

  for (int i = 0; i < info->num_controls; i++) {
    int idx = 0;
    while (info->permutation[idx] != info->target[i])
      idx++;
    controls.push_back(idx);
  }

  // apply gate
  custatevecApplyMatrix(
      /* custatevecHandle_t */ handler->statevec,
      /* void* */ state_vector.get_void_ptr(),
      /* hipDataType */ data_type,
      /* const uint32_t */ nIndexBits,
      /* const void* */ info->matrix_data,
      /* hipDataType */ data_type,
      /* custatevecMatrixLayout_t */ CUSTATEVEC_MATRIX_LAYOUT_ROW,
      /* const int32_t */ adjoint,
      /* const int32_t* */ targets.data(),
      /* const uint32_t */ nTargets,
      /* const int32_t* */ controls.data(),
      /* const int32_t* */ nullptr,
      /* const uint32_t */ nControls,
      /* custatevecComputeType_t */ compute_type,
      /* void* */ handler->workSpace,
      /* size_t */ handler->workSpaceSize);
}
